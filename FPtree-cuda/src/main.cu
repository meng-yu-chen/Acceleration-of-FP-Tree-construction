#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <set>
#include <vector>
#include <fstream>
#include <string>
#include <filesystem>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iomanip> 
#include <hip/hip_runtime.h>
#include <sstream>
#include <stdexcept>
#define CATCH_CONFIG_RUNNER
#include "catch.hpp"

#include "FPGrowth.h"
#include "FPHeaderTable.h"
#include "FPRadixTree.h"
#include "FPTransMap.h"

#include <thrust/device_vector.h>
#include <thrust/extrema.h>
// include timer
#include "CycleTimer.h"


using namespace std;
using namespace cuda_fp_growth;

void load_transactions_from_file(const std::string& filename, Items& trans, Indices& indices, Sizes& sizes) {
    std::ifstream infile(filename);
    if (!infile.is_open()) {
        throw std::runtime_error("Failed to open file for reading: " + filename);
    }

    std::string line;
    size_t current_index = 0; // 用來記錄當前交易的起始索引

    while (std::getline(infile, line)) {
        std::istringstream iss(line);
        std::string item_str;
        size_type line_item_count = 0; // 記錄當前交易的項目數量

        // 記錄當前交易的起始索引
        indices.push_back(static_cast<index_type>(current_index));

        // 解析當前行的所有項目
        while (std::getline(iss, item_str, ',')) {
            // 將數字轉為 `Item` 並檢查範圍
            Item item = static_cast<Item>(std::stoul(item_str));
            if (item > 100) {
                throw std::runtime_error("Item value out of range (0-100): " + item_str);
            }
            trans.push_back(item);
            line_item_count++;
        }

        // 記錄當前交易的項目數量
        sizes.push_back(line_item_count);

        // 更新下一交易的起始索引
        current_index += line_item_count;
    }

    infile.close();
}

size_type pattern_count( const std::vector<cuda_uint>& buffer )
{
    index_type i = 0;
    size_type pattern_count = 0;
    while ( i < buffer.size() ) {
        ++pattern_count;
        i += ( buffer[i] / sizeof( cuda_uint ) );
    }
    return pattern_count;
}

bool pattern_exists( const std::vector<cuda_uint>& buffer, const std::vector<Item>& pattern, const size_type support, const cuda_real confidence = 0.0f )
{
    index_type i = 0;
    while ( i < buffer.size() ) {
        size_type length = buffer[ i ] / sizeof( cuda_uint );
        size_type offset = ( confidence > 0.0f ? 3 : 2 );
        bool exists = true;
        exists &= ( pattern.size() == length - offset );
        exists &= ( buffer[ i + 1 ] == support );
        exists &= ( std::equal( pattern.begin(), pattern.end(), buffer.begin() + i + offset ) );
        if ( confidence > 0.0f ) {
            const cuda_uint* ptr = &buffer[ i + 2 ];
            exists &= ( std::abs( *( reinterpret_cast<const cuda_real*>( ptr ) ) - confidence ) < 0.0001 );
        }
        if ( exists ) return true;

        i += ( buffer[i] / sizeof( cuda_uint ) );
    }
    return false;
}

int main(int argc, const char *argv[])
{
    string data_folderPath = argv[1];
    
    Items trans;      // 用來存放所有項目
    Indices indices;  // 每筆交易的起始索引
    Sizes sizes;      // 每筆交易的項目數量

    load_transactions_from_file(data_folderPath, trans, indices, sizes);

    double start_time = CycleTimer::currentSeconds();

    //step 1 :建立 FPTransMap 物件
    size_type min_support = 3;
    std::cout << "Testing FPTransMap with minimum support = 3 \n";
    FPTransMap fp_trans_map( trans.cbegin(), indices.cbegin(), sizes.cbegin(), indices.size(), min_support );

    // 取得頻繁項目
    const DItems& d_freq_items = fp_trans_map.frequent_items();
    const DSizes& d_freq_items_counts = fp_trans_map.items_frequency();
    const Items freq_items( d_freq_items.begin(), d_freq_items.end() );
    const Sizes freq_items_counts( d_freq_items_counts.begin(), d_freq_items_counts.end() );

    //step 2 :FPRadixTree
    FPRadixTree fp_radix_tree( fp_trans_map );
    const DInnerNodes& d_inner_nodes = fp_radix_tree.inner_nodes();
    const DLeafNodes& d_leaf_nodes = fp_radix_tree.leaf_nodes();
    InnerNodes inner_nodes(d_inner_nodes.cbegin(), d_inner_nodes.cend());
    LeafNodes leaf_nodes(d_leaf_nodes.cbegin(), d_leaf_nodes.cend());
    
    //step 3 :FPHeaderTable
    FPHeaderTable header_table( fp_trans_map, fp_radix_tree, min_support );

    double exec_time = CycleTimer::currentSeconds() - start_time;
    cout << "Test case - " << data_folderPath << " time : " << setprecision(4) << exec_time <<"s\n";

    //step 4 :FPGrowth
    FPGrowth fp( fp_trans_map, fp_radix_tree, header_table, min_support );
    
    std::vector<cuda_uint> buffer( 1024 );
    size_type buffer_size = sizeof( cuda_uint ) * buffer.size();

    fp.mine_frequent_patterns( &buffer[0], buffer_size );

    buffer.resize( buffer_size / sizeof( cuda_uint ) );
        
    // test結果
    if(data_folderPath == "./dataset/test.txt"){

        // 顯示載入的結果        
        std::cout << "Items (trans): ";
        for (const auto& item : trans) {
            std::cout << item << " ";
        }
        std::cout << "\n";

        std::cout << "Indices: ";
        for (const auto& index : indices) {
            std::cout << index << " ";
        }
        std::cout << "\n";

        std::cout << "Sizes: ";
        for (const auto& size : sizes) {
            std::cout << size << " ";
        }
        std::cout << "\n";

        // 驗證頻繁項目的正確性
        std::cout << "freq_items.size(): " << freq_items.size() << std::endl;
        std::cout << "freq_items_counts.size(): " << freq_items_counts.size() << std::endl;
        std::cout << "Frequent items and counts are correctly identified.\n";

        // 顯示頻繁項目及其支持數
        for (size_t i = 0; i < freq_items.size(); ++i) {
            std::cout << "Item: " << freq_items[i] << ", Count: " << freq_items_counts[i] << "\n";
        }

        // 驗證內部節點和葉節點的非空性（或其他必要條件）
        std::cout << "!inner_nodes.empty(): " << !inner_nodes.empty() << std::endl;
        std::cout << "!leaf_nodes.empty(): " << !leaf_nodes.empty() << std::endl;
        std::cout << "FPRadixTree inner nodes and leaf nodes are correctly constructed.\n";

        // 顯示內部節點資訊 
        std::cout << "Inner Nodes:\n";
        for (const auto& node : inner_nodes) {
            std::cout << "Parent: " << node.parent_idx
                      << ", Range: [" << node.range_start << ", " << node.range_end << "]"
                      << ", Left: " << (node.left_is_leaf ? "Leaf" : "Inner") << " (" << node.left_idx << ")"
                      << ", Right: " << (node.right_is_leaf ? "Leaf" : "Inner") << " (" << node.right_idx << ")"
                      << ", Prefix Length: " << node.prefix_length
                      << ", Transaction Count: " << node.trans_count << "\n";
        }

        // 顯示葉節點資訊   
        std::cout << "Leaf Nodes:\n";
        for (const auto& node : leaf_nodes) {
            std::cout << "Parent: " << node.parent_idx
                      << ", Transaction Count: " << node.trans_count << "\n";
        }

        // 測試 FPHeaderTable
        //hipDeviceSynchronize();
        std::cout << "Header Table Size: " << header_table.size() << std::endl;
        std::cout << "header_table.ia_size: " << header_table.ia_size() << std::endl;

    }

    return EXIT_SUCCESS;
}
