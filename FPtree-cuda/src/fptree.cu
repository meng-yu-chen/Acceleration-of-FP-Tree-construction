#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdint>
#include <utility>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/set_operations.h>
#include <set>
#include <vector>
#include <map>
#include <string>
#include <cassert>

#include "fptree.hpp"
#include "CycleTimer.h"

FPNode::FPNode(const Item& item, const std::shared_ptr<FPNode>& parent) :
    item( item ), frequency( 1 ), node_link( nullptr ), parent( parent ), children()
{
}

__global__ void calculate_frequencies(const int* d_flattened_transactions, 
                                      int num_transactions, 
                                      int* d_frequency, 
                                      int flattened_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < flattened_size) {
        atomicAdd(&d_frequency[d_flattened_transactions[idx]], 1);
    }
}

__global__ void filter_items(const int* d_frequency, 
                             int* d_filtered_items, 
                             int num_items, 
                             int minimum_support_threshold) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_items) {
        d_filtered_items[idx] = (d_frequency[idx] >= minimum_support_threshold) ? 1 : 0;
    }
}

FPTree::FPTree(const std::vector<Transaction>& transactions, uint64_t minimum_support_threshold) :
    root( std::make_shared<FPNode>( Item{}, nullptr ) ), header_table(),
    minimum_support_threshold( minimum_support_threshold )
{
   //static int what = 0;
    // Flatten transactions and prepare offsets
    std::map<Item, int> item_to_index;
    std::vector<int> flattened_transactions;
    
    int current_index = 0;

    double startTime = CycleTimer::currentSeconds();

    for (const auto& transaction : transactions) {
        for (const auto& item : transaction) {
            if (item_to_index.find(item) == item_to_index.end()) {
                item_to_index[item] = current_index++;
            }
            flattened_transactions.push_back(item_to_index[item]);
        }
    }

    int num_items = item_to_index.size();
    int flattened_size = flattened_transactions.size();

    // Allocate CUDA memory
    int* d_flattened_transactions;
    int* d_frequency;
    int* d_filtered_items;
    hipMalloc(&d_flattened_transactions, flattened_size * sizeof(int));
    hipMalloc(&d_frequency, num_items * sizeof(int));
    hipMalloc(&d_filtered_items, num_items * sizeof(int));
    hipMemset(d_frequency, 0, num_items * sizeof(int));

    // Copy data to device
    hipMemcpy(d_flattened_transactions, flattened_transactions.data(), flattened_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch frequency calculation kernel
    int blockSize = 256;
    int numBlocks = (flattened_size + blockSize - 1) / blockSize;
    calculate_frequencies<<<numBlocks, blockSize>>>(d_flattened_transactions, 
                                                    transactions.size(), 
                                                    d_frequency, 
                                                    flattened_size);
    hipDeviceSynchronize();

    // Launch filtering kernel
    numBlocks = (num_items + blockSize - 1) / blockSize;
    filter_items<<<numBlocks, blockSize>>>(d_frequency, 
                                           d_filtered_items, 
                                           num_items, 
                                           minimum_support_threshold);
    hipDeviceSynchronize();

    // Copy filtered results back to host
    std::vector<int> h_filtered_items(num_items);
    hipMemcpy(h_filtered_items.data(), d_filtered_items, num_items * sizeof(int), hipMemcpyDeviceToHost);

    // Create frequency_by_item map
    std::map<Item, uint64_t> frequency_by_item;
    for (const auto& [item, index] : item_to_index) {
        if (h_filtered_items[index]) {
            frequency_by_item[item] = h_filtered_items[index];
        }
    }

    // Clean up
    hipFree(d_flattened_transactions);
    hipFree(d_frequency);
    hipFree(d_filtered_items);

    double endTime = CycleTimer::currentSeconds();
    
    //what++;
    //printf("cuda time %d = %lf s\n", what, endTime - startTime);


    // Order items by decreasing frequency (same as original code)
    struct frequency_comparator {
        bool operator()(const std::pair<Item, uint64_t>& lhs, const std::pair<Item, uint64_t>& rhs) const {
            return std::tie(lhs.second, lhs.first) > std::tie(rhs.second, rhs.first);
        }
    };

    std::set<std::pair<Item, uint64_t>, frequency_comparator> items_ordered_by_frequency(frequency_by_item.cbegin(), frequency_by_item.cend());

    // start tree construction

    // scan the transactions again
    for ( const Transaction& transaction : transactions ) {
        auto curr_fpnode = root;

        // select and sort the frequent items in transaction according to the order of items_ordered_by_frequency
        for ( const auto& pair : items_ordered_by_frequency ) {
            const Item& item = pair.first;

            // check if item is contained in the current transaction
            if ( std::find( transaction.cbegin(), transaction.cend(), item ) != transaction.cend() ) {
                // insert item in the tree

                // check if curr_fpnode has a child curr_fpnode_child such that curr_fpnode_child.item = item
                const auto it = std::find_if(
                    curr_fpnode->children.cbegin(), curr_fpnode->children.cend(),  [item](const std::shared_ptr<FPNode>& fpnode) {
                        return fpnode->item == item;
                } );
                if ( it == curr_fpnode->children.cend() ) {
                    // the child doesn't exist, create a new node
                    const auto curr_fpnode_new_child = std::make_shared<FPNode>( item, curr_fpnode );

                    // add the new node to the tree
                    curr_fpnode->children.push_back( curr_fpnode_new_child );

                    // update the node-link structure
                    if ( header_table.count( curr_fpnode_new_child->item ) ) {
                        auto prev_fpnode = header_table[curr_fpnode_new_child->item];
                        while ( prev_fpnode->node_link ) { prev_fpnode = prev_fpnode->node_link; }
                        prev_fpnode->node_link = curr_fpnode_new_child;
                    }
                    else {
                        header_table[curr_fpnode_new_child->item] = curr_fpnode_new_child;
                    }

                    // advance to the next node of the current transaction
                    curr_fpnode = curr_fpnode_new_child;
                }
                else {
                    // the child exist, increment its frequency
                    auto curr_fpnode_child = *it;
                    ++curr_fpnode_child->frequency;

                    // advance to the next node of the current transaction
                    curr_fpnode = curr_fpnode_child;
                }
            }
        }
    }
}



bool FPTree::empty() const
{
    assert( root );
    return root->children.size() == 0;
}


bool contains_single_path(const std::shared_ptr<FPNode>& fpnode)
{
    assert( fpnode );
    if ( fpnode->children.size() == 0 ) { return true; }
    if ( fpnode->children.size() > 1 ) { return false; }
    return contains_single_path( fpnode->children.front() );
}
bool contains_single_path(const FPTree& fptree)
{
    return fptree.empty() || contains_single_path( fptree.root );
}



std::set<Pattern> fptree_growth(const FPTree& fptree)
{
    if ( fptree.empty() ) { return {}; }

    if ( contains_single_path( fptree ) ) {
        // generate all possible combinations of the items in the tree

        std::set<Pattern> single_path_patterns;

        // for each node in the tree
        assert( fptree.root->children.size() == 1 );
        auto curr_fpnode = fptree.root->children.front();
        while ( curr_fpnode ) {
            const Item& curr_fpnode_item = curr_fpnode->item;
            const uint64_t curr_fpnode_frequency = curr_fpnode->frequency;

            // add a pattern formed only by the item of the current node
            Pattern new_pattern{ { curr_fpnode_item }, curr_fpnode_frequency };
            single_path_patterns.insert( new_pattern );

            // create a new pattern by adding the item of the current node to each pattern generated until now
            for ( const Pattern& pattern : single_path_patterns ) {
                Pattern new_pattern{ pattern };
                new_pattern.first.insert( curr_fpnode_item );
                assert( curr_fpnode_frequency <= pattern.second );
                new_pattern.second = curr_fpnode_frequency;

                single_path_patterns.insert( new_pattern );
            }

            // advance to the next node until the end of the tree
            assert( curr_fpnode->children.size() <= 1 );
            if ( curr_fpnode->children.size() == 1 ) { curr_fpnode = curr_fpnode->children.front(); }
            else { curr_fpnode = nullptr; }
        }

        return single_path_patterns;
    }
    else {
        // generate conditional fptrees for each different item in the fptree, then join the results

        std::set<Pattern> multi_path_patterns;

        // for each item in the fptree
        for ( const auto& pair : fptree.header_table ) {
            const Item& curr_item = pair.first;

            // build the conditional fptree relative to the current item

            // start by generating the conditional pattern base
            std::vector<TransformedPrefixPath> conditional_pattern_base;

            // for each path in the header_table (relative to the current item)
            auto path_starting_fpnode = pair.second;
            while ( path_starting_fpnode ) {
                // construct the transformed prefix path

                // each item in th transformed prefix path has the same frequency (the frequency of path_starting_fpnode)
                const uint64_t path_starting_fpnode_frequency = path_starting_fpnode->frequency;

                auto curr_path_fpnode = path_starting_fpnode->parent.lock();
                // check if curr_path_fpnode is already the root of the fptree
                if ( curr_path_fpnode->parent.lock() ) {
                    // the path has at least one node (excluding the starting node and the root)
                    TransformedPrefixPath transformed_prefix_path{ {}, path_starting_fpnode_frequency };

                    while ( curr_path_fpnode->parent.lock() ) {
                        assert( curr_path_fpnode->frequency >= path_starting_fpnode_frequency );
                        transformed_prefix_path.first.push_back( curr_path_fpnode->item );

                        // advance to the next node in the path
                        curr_path_fpnode = curr_path_fpnode->parent.lock();
                    }

                    conditional_pattern_base.push_back( transformed_prefix_path );
                }

                // advance to the next path
                path_starting_fpnode = path_starting_fpnode->node_link;
            }

            // generate the transactions that represent the conditional pattern base
            std::vector<Transaction> conditional_fptree_transactions;
            for ( const TransformedPrefixPath& transformed_prefix_path : conditional_pattern_base ) {
                const std::vector<Item>& transformed_prefix_path_items = transformed_prefix_path.first;
                const uint64_t transformed_prefix_path_items_frequency = transformed_prefix_path.second;

                Transaction transaction = transformed_prefix_path_items;

                // add the same transaction transformed_prefix_path_items_frequency times
                for ( auto i = 0; i < transformed_prefix_path_items_frequency; ++i ) {
                    conditional_fptree_transactions.push_back( transaction );
                }
            }

            // build the conditional fptree relative to the current item with the transactions just generated
            const FPTree conditional_fptree( conditional_fptree_transactions, fptree.minimum_support_threshold );
            // call recursively fptree_growth on the conditional fptree (empty fptree: no patterns)
            std::set<Pattern> conditional_patterns = fptree_growth( conditional_fptree );

            // construct patterns relative to the current item using both the current item and the conditional patterns
            std::set<Pattern> curr_item_patterns;

            // the first pattern is made only by the current item
            // compute the frequency of this pattern by summing the frequency of the nodes which have the same item (follow the node links)
            uint64_t curr_item_frequency = 0;
            auto fpnode = pair.second;
            while ( fpnode ) {
                curr_item_frequency += fpnode->frequency;
                fpnode = fpnode->node_link;
            }
            // add the pattern as a result
            Pattern pattern{ { curr_item }, curr_item_frequency };
            curr_item_patterns.insert( pattern );

            // the next patterns are generated by adding the current item to each conditional pattern
            for ( const Pattern& pattern : conditional_patterns ) {
                Pattern new_pattern{ pattern };
                new_pattern.first.insert( curr_item );
                assert( curr_item_frequency >= pattern.second );
                new_pattern.second = pattern.second;

                curr_item_patterns.insert( { new_pattern } );
            }

            // join the patterns generated by the current item with all the other items of the fptree
            multi_path_patterns.insert( curr_item_patterns.cbegin(), curr_item_patterns.cend() );
        }

        return multi_path_patterns;
    }
}


